#include<stdio.h>
#include<hip/hip_runtime.h>
#include<stdlib.h>
#include<sys/time.h>
#include<time.h>

#define STREAM_NUMBER 10

//Macro for checking cuda errors following a cuda launch or api call
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }

void initialize(int *a, int *b, int *c, int *d, int input_length) {
	for (int i = 0; i < input_length; i++) {
		a[i] = rand() % 100;
		b[i] = rand() % 100;
		c[i] = 0;
		d[i] = 0;
	}
}

void validate(int *a, int *b, int length) {
	for (int i = 0; i < length; ++i) {
		if (a[i] != b[i]) {
			printf("Different value detected at position: %d,"
					"expected %d but get %d\n", i, a[i], b[i]);
			break;
		}
	}
}

void vector_add(int *a, int *b, int *c, int size) {
	for (int i = 0; i < size; i++) {
		c[i] = a[i] + b[i];
	}
}

__global__ void vector_add_kernel_coalesced_access(int *a_d, int *b_d, int *d_d, int work_per_thread, int input_length, int totalThreads) {

	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	for(int i = 0; i < work_per_thread && tid < input_length; i++, tid += totalThreads){
		d_d[tid] = a_d[tid] + b_d[tid];
	}

}

int main(int argc, char *argv[]) {

	int input_length, block_size, work_per_thread;
	struct timeval start, end;
	hipStream_t streams[STREAM_NUMBER];


	if (argc != 2) {
		printf("Usage is: VectorAddParallel input_length block_size work_per_thread\nNow, type input_length: ");
		scanf("%d", &input_length);
		printf("Type block_size: ");
		scanf("%d", &block_size);
		printf("Type work_per_thread: ");
		scanf("%d", &work_per_thread);
	} else{
		input_length = atoi(argv[1]);
		block_size = atoi(argv[2]);
		work_per_thread = atoi(argv[3]);
	}

	// Arrays declaration
	int *a_h, *b_h, *c_h, *d_h;
	int *a_d, *b_d, *d_d;

	// Allocation on Host
	//a_h = (int *) malloc(sizeof(int) * input_length);
	//b_h = (int *) malloc(sizeof(int) * input_length);
	c_h = (int *) malloc(sizeof(int) * input_length);
	//d_h = (int *) malloc(sizeof(int) * input_length);

	hipHostMalloc(&a_h, sizeof(int) * input_length, hipHostMallocDefault);
	hipHostMalloc(&b_h, sizeof(int) * input_length, hipHostMallocDefault);
	//cudaMallocHost(&c_h, sizeof(int) * input_length);
	hipHostMalloc(&d_h, sizeof(int) * input_length, hipHostMallocDefault);
	
	// Allocation on Device
	CUDA_CHECK_RETURN(hipMalloc((void **)&a_d, sizeof(int)*input_length));
	CUDA_CHECK_RETURN(hipMalloc((void **)&b_d, sizeof(int)*input_length));
	CUDA_CHECK_RETURN(hipMalloc((void **)&d_d, sizeof(int)*input_length));

	// Initialization on host side
	initialize(a_h, b_h, c_h, d_h, input_length);

	// Run host code
	gettimeofday(&start, NULL);
	vector_add(a_h, b_h, c_h, input_length);
	gettimeofday(&end, NULL);

	double diff = (end.tv_sec - start.tv_sec) * 1000000.0 +
		(end.tv_usec - start.tv_usec);

	printf("Host VectorAdd time calculation duration: %8.5fms\n", diff / 1000);

	// Run device code

	
	for (int i = 0; i < STREAM_NUMBER; ++i) {
		hipStreamCreate(&streams[i]);
	}

	gettimeofday(&start, NULL);

	// Copy input data to device
	//CUDA_CHECK_RETURN(cudaMemcpy(a_d, a_h, sizeof(int)*input_length, cudaMemcpyHostToDevice));
	//CUDA_CHECK_RETURN(cudaMemcpy(b_d, b_h, sizeof(int)*input_length, cudaMemcpyHostToDevice));

	int streamSize = input_length / STREAM_NUMBER;
	int streamBytes = streamSize * 4;
	
	int grid_size = (streamSize - 1) / (block_size * work_per_thread) + 1;
	dim3 grid_dime(grid_size, 1, 1);
	dim3 block_dime(block_size, 1, 1);
	int totalThreads = grid_size * block_size;

	for (int i = 0; i < STREAM_NUMBER + 2; ++i) {
		int offset = i * streamSize;

		if(i < STREAM_NUMBER){
			hipMemcpyAsync(&a_d[offset], &a_h[offset], streamBytes, hipMemcpyHostToDevice, streams[i]);
			hipMemcpyAsync(&b_d[offset], &b_h[offset], streamBytes, hipMemcpyHostToDevice, streams[i]);
		}
		if(i > 0 && i < STREAM_NUMBER + 1){
			offset = (i-1) * streamSize;
			vector_add_kernel_coalesced_access<<< grid_dime, block_dime, 0, streams[i-1]>>>(&a_d[offset], &b_d[offset], &d_d[offset], work_per_thread, streamSize, totalThreads);
		}
		if(i > 1){
			offset = (i-2) * streamSize;
			hipMemcpyAsync(&d_h[offset], &d_d[offset], streamBytes, hipMemcpyDeviceToHost, streams[i-2]);
		}
	}
	
	//Copy back the result
	//CUDA_CHECK_RETURN(cudaMemcpy(d_h, d_d, sizeof(int)*input_length, cudaMemcpyDeviceToHost));

	CUDA_CHECK_RETURN(hipDeviceSynchronize());	// Wait for the GPU launched work to complete
	CUDA_CHECK_RETURN(hipGetLastError());
	gettimeofday(&end, NULL);

	diff = (end.tv_sec - start.tv_sec) * 1000000.0 +
		(end.tv_usec - start.tv_usec);

	printf("Device VectorAdd time calculation duration: %8.5fms\n", diff / 1000);

	// Validation
	validate(c_h, d_h, input_length);

	hipFree(a_h);
	hipFree(b_h);
	free(c_h);
	hipFree(d_h);
	hipFree(a_d);
	hipFree(b_d);
	hipFree(d_d);

	return 0;
}
