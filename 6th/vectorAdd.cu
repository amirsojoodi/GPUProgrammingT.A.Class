#include<stdio.h>
#include<hip/hip_runtime.h>
#include<stdlib.h>
#include<sys/time.h>
#include<time.h>

//Macro for checking cuda errors following a cuda launch or api call
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }

void initialize(int *a, int *b, int *c, int *d, int input_length) {
	for (int i = 0; i < input_length; i++) {
		a[i] = rand() % 100;
		b[i] = rand() % 100;
		c[i] = 0;
		d[i] = 0;
	}
}

void validate(int *a, int *b, int length) {
    for (int i = 0; i < length; ++i) {
        if (a[i] != b[i]) {
            printf("Different value detected at position: %d,"
                    "expected %d but get %d\n", i, a[i], b[i]);
            break;
        }
    }
}

void vector_add(int *a, int *b, int *c, int size) {
    for (int i = 0; i < size; i++) {
        c[i] = a[i] + b[i];
    }
}

__global__ void vector_add_kernel(int *a_d, int *b_d, int *d_d, int work_per_thread, int input_length) {
    
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	int start = tid * work_per_thread;
	int end = start + work_per_thread;

	for(int i = start; i < end && i < input_length; i++){
		d_d[i] = a_d[i] + b_d[i];
	}

}

int main(int argc, char *argv[]) {

    int input_length, block_size, work_per_thread;
	struct timeval start, end;

    if (argc != 2) {
        printf("Usage is: VectorAddParallel input_length block_size work_per_thread\nNow, type input_length: ");
        scanf("%d", &input_length);
		printf("Type block_size: ");
		scanf("%d", &block_size);
		printf("Type work_per_thread: ");
		scanf("%d", &work_per_thread);
    } else{
        input_length = atoi(argv[1]);
		block_size = atoi(argv[2]);
		work_per_thread = atoi(argv[3]);
    }

	// Arrays declaration
    int *a_h, *b_h, *c_h, *d_h;
	int *a_d, *b_d, *d_d;

	// Allocation on Host
    a_h = (int *) malloc(sizeof(int) * input_length);
    b_h = (int *) malloc(sizeof(int) * input_length);
    c_h = (int *) malloc(sizeof(int) * input_length);
    d_h = (int *) malloc(sizeof(int) * input_length);

	// Allocation on Device
	CUDA_CHECK_RETURN(hipMalloc((void **)&a_d, sizeof(int)*input_length));
	CUDA_CHECK_RETURN(hipMalloc((void **)&b_d, sizeof(int)*input_length));
	CUDA_CHECK_RETURN(hipMalloc((void **)&d_d, sizeof(int)*input_length));

	// Initialization on host side
    initialize(a_h, b_h, c_h, d_h, input_length);

	// Run host code
	gettimeofday(&start, NULL);
	vector_add(a_h, b_h, c_h, input_length);
	gettimeofday(&end, NULL);
	
	double diff = (end.tv_sec - start.tv_sec) * 1000000.0 +
        (end.tv_usec - start.tv_usec);

    printf("Host VectorAdd time calculation duration: %8.5fms\n", diff / 1000);

	// Run device code
	int grid_size = (input_length - 1) / (block_size * work_per_thread) + 1;
		
	dim3 grid_dime(grid_size, 1, 1);
	dim3 block_dime(block_size, 1, 1);

	gettimeofday(&start, NULL);
	
	// Copy input data to device
	CUDA_CHECK_RETURN(hipMemcpy(a_d, a_h, sizeof(int)*input_length, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(b_d, b_h, sizeof(int)*input_length, hipMemcpyHostToDevice));

	vector_add_kernel<<< grid_dime, block_dime >>>(a_d, b_d, d_d, work_per_thread, input_length);

	CUDA_CHECK_RETURN(hipDeviceSynchronize());	// Wait for the GPU launched work to complete
	CUDA_CHECK_RETURN(hipGetLastError());
	
	//Copy back the result
	CUDA_CHECK_RETURN(hipMemcpy(d_h, d_d, sizeof(int)*input_length, hipMemcpyDeviceToHost));

	gettimeofday(&end, NULL);

	diff = (end.tv_sec - start.tv_sec) * 1000000.0 +
        (end.tv_usec - start.tv_usec);
        
    printf("Device VectorAdd time calculation duration: %8.5fms\n", diff / 1000);

	// Validation
	validate(c_h, d_h, input_length);

	free(a_h);
	free(b_h);
	free(c_h);
	free(d_h);
	hipFree(a_d);
	hipFree(b_d);
	hipFree(d_d);

	return 0;
}
